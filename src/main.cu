#include <iostream>
#include <fstream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include "utils.h"

int main(int argc, char *argv[]) {
    if (argc < 2) {
        std::cerr << "Usage: ./fft_analyzer <input_csv>\n";
        return 1;
    }

    std::string input_file = argv[1];
    std::vector<float> signal = readCSV(input_file);
    int N = signal.size();

    hipfftComplex *d_data;
    hipMalloc((void **)&d_data, sizeof(hipfftComplex) * N);

    std::vector<hipfftComplex> complex_input(N);
    for (int i = 0; i < N; ++i) {
        complex_input[i].x = signal[i];
        complex_input[i].y = 0;
    }

    hipMemcpy(d_data, complex_input.data(), sizeof(hipfftComplex) * N, hipMemcpyHostToDevice);

    hipfftHandle plan;
    hipfftPlan1d(&plan, N, HIPFFT_C2C, 1);
    hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD);

    hipMemcpy(complex_input.data(), d_data, sizeof(hipfftComplex) * N, hipMemcpyDeviceToHost);
    saveFFTOutput("results/" + getBaseName(input_file) + "_fft_output.csv", complex_input);

    hipfftDestroy(plan);
    hipFree(d_data);

    std::cout << "FFT complete for " << input_file << "\n";
    return 0;
}
